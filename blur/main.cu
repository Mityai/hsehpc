#include "hip/hip_runtime.h"
#include <cassert>
#include <chrono>
#include <cmath>
#include <cstdio>
#include <cstdlib>
#include <hip/hip_runtime.h>
#include <iostream>
#include <limits>

#define IDX(i, j, m) ((i) * (m) + (j))

using uchar = unsigned char;

const size_t BLOCK_SIZE = 16;

const int BLUR_KERNEL_WIDTH = 9;
const float BLUR_KERNEL_SIGMA = 4;
const float PI = acos(-1);

template <typename T>
void allocate_on_gpu(T** data, size_t n, size_t m) {
  hipMalloc(data, sizeof(T) * n * m);
}

template <typename T>
void memset_on_gpu(T* data, size_t n, size_t m) {
  hipMemset(data, 0, sizeof(T) * n * m);
}

template <typename T>
void copy_to_gpu(const T* const from, T* to, size_t n, size_t m) {
  hipMemcpy(to, from, sizeof(T) * n * m, hipMemcpyHostToDevice);
}

template <typename T>
void copy_to_host(const T* const from, T* to, size_t n, size_t m) {
  hipMemcpy(to, from, sizeof(T) * n * m, hipMemcpyDeviceToHost);
}

template <typename T>
void free_on_gpu(T* data) {
  hipFree(data);
}

template <typename T>
void allocate(T** data, size_t n, size_t m) {
  *data = (T*)malloc(sizeof(T) * n * m);
}

template <typename T>
T sqr(T x) {
  return x * x;
}

void read_rgb(size_t height, size_t width, size_t max_height, size_t max_width,
               uchar** red, uchar** green, uchar** blue)
{
   for (uchar** channel : {red, green, blue}) {
     allocate(channel, min(height, max_height), min(width, max_width));
     for (size_t i = 0; i < height; ++i) {
       for (size_t j = 0; j < width; ++j) {
         int value;
         std::cin >> value;
         if (i < max_height && j < max_width) {
           (*channel)[IDX(i, j, max_width)] = value;
         }
       }
     }
   }
}

void get_filter_gpu(float** device_filter) {
  float* filter;
  allocate(&filter, BLUR_KERNEL_WIDTH, BLUR_KERNEL_WIDTH);
  const int half_width = BLUR_KERNEL_WIDTH / 2;

  float sum = 0;
  for (int i = -half_width; i <= half_width; ++i) {
    for (int j = -half_width; j <= half_width; ++j) {
      const float value = expf(-(sqr(i) + sqr(j)) / (2 * sqr(BLUR_KERNEL_SIGMA))) / (2 * PI * sqr(BLUR_KERNEL_SIGMA));
      filter[IDX(i + half_width, j + half_width, BLUR_KERNEL_WIDTH)] = value;
      sum += value;
    }
  }

  for (int i = -half_width; i <= half_width; ++i) {
    for (int j = -half_width; j <= half_width; ++j) {
      filter[IDX(i + half_width, j + half_width, BLUR_KERNEL_WIDTH)] /= sum;
    }
  }

  allocate_on_gpu(device_filter, BLUR_KERNEL_WIDTH, BLUR_KERNEL_WIDTH);
  copy_to_gpu(filter, *device_filter, BLUR_KERNEL_WIDTH, BLUR_KERNEL_WIDTH);
}

__device__ size_t get_pos(int initial_pos, int delta, int bound) {
  int result = initial_pos + delta - BLUR_KERNEL_WIDTH / 2;
  if (result < 0) return 0;
  if (result >= bound) return bound - 1;
  return result;
}

__global__ void gaussian_blur(const uchar* const input, uchar* const output, size_t height, size_t width, const float* const filter) {
  const size_t col = blockIdx.x * blockDim.x + threadIdx.x;
  const size_t row = blockIdx.y * blockDim.y + threadIdx.y;
  if (row >= height || col >= width) {
    return;
  }

  float value = 0.0;
  for (size_t i = 0; i < BLUR_KERNEL_WIDTH; ++i) {
    for (size_t j = 0; j < BLUR_KERNEL_WIDTH; ++j) {
      const size_t image_row = get_pos(row, i, height);
      const size_t image_col = get_pos(col, j, width);
      value += filter[IDX(i, j, BLUR_KERNEL_WIDTH)] * input[IDX(image_row, image_col, width)];
    }
  }

  output[IDX(row, col, width)] = value;  
}

void blur_image_gpu_channel(size_t height, size_t width, const uchar* const input_matrix, const float* const filter, uchar** device_output) {
  uchar* device_input;
  allocate_on_gpu(&device_input, height, width);
  copy_to_gpu(input_matrix, device_input, height, width);

  allocate_on_gpu(device_output, height, width);
  memset_on_gpu(*device_output, height, width);

  const dim3 dim_block(BLOCK_SIZE, BLOCK_SIZE);
  const dim3 dim_grid((width + dim_block.x - 1) / dim_block.x, (height + dim_block.y - 1) / dim_block.y);

  gaussian_blur<<<dim_grid, dim_block>>>(device_input, *device_output, height, width, filter);
  free_on_gpu(device_input);
}

void blur_image_channel(size_t height, size_t width, const uchar* const input, const float* const filter, uchar** output) {
  uchar* device_blured;
  blur_image_gpu_channel(height, width, input, filter, &device_blured);

  hipDeviceSynchronize();

  allocate(output, height, width);
  copy_to_host(device_blured, *output, height, width);

  free_on_gpu(device_blured);
}

void blur_image_gpu(size_t height, size_t width, const uchar* const input_red, const uchar* const input_green, const uchar* const input_blue,
                    uchar** output_red, uchar** output_green, uchar** output_blue)
{
  float* device_filter;
  get_filter_gpu(&device_filter);

  hipDeviceSynchronize();

  blur_image_channel(height, width, input_red, device_filter, output_red);
  blur_image_channel(height, width, input_green, device_filter, output_green);
  blur_image_channel(height, width, input_blue, device_filter, output_blue);
}

void get_filter_cpu(float** filter) {
  allocate(filter, BLUR_KERNEL_WIDTH, BLUR_KERNEL_WIDTH);
  const int half_width = BLUR_KERNEL_WIDTH / 2;

  float sum = 0;
  for (int i = -half_width; i <= half_width; ++i) {
    for (int j = -half_width; j <= half_width; ++j) {
      const float value = expf(-(sqr(i) + sqr(j)) / (2 * sqr(BLUR_KERNEL_SIGMA))) / (2 * PI * sqr(BLUR_KERNEL_SIGMA));
      (*filter)[IDX(i + half_width, j + half_width, BLUR_KERNEL_WIDTH)] = value;
      sum += value;
    }
  }

  for (int i = -half_width; i <= half_width; ++i) {
    for (int j = -half_width; j <= half_width; ++j) {
      (*filter)[IDX(i + half_width, j + half_width, BLUR_KERNEL_WIDTH)] /= sum;
    }
  }
}

void blur_image_cpu_channel(size_t height, size_t width, const uchar* const input, const float* const filter, uchar** output) {
  allocate(output, height, width);

  auto get_pos = [](int initial_pos, int delta, int bound) -> int {
    int result = initial_pos + delta - BLUR_KERNEL_WIDTH / 2;
    if (result < 0) return 0;
    if (result >= bound) return bound - 1;
    return result;
  };

  for (size_t row = 0; row < height; ++row) {
    for (size_t col = 0; col < width; ++col) {
      float value = 0.0;
      for (size_t i = 0; i < BLUR_KERNEL_WIDTH; ++i) {
        for (size_t j = 0; j < BLUR_KERNEL_WIDTH; ++j) {
          const size_t image_row = get_pos(row, i, height);
          const size_t image_col = get_pos(col, j, width);
          value += filter[IDX(i, j, BLUR_KERNEL_WIDTH)] * input[IDX(image_row, image_col, width)];
        }
      }
      (*output)[IDX(row, col, width)] = value;
    }
  }
}

void blur_image_cpu(size_t height, size_t width, const uchar* const input_red, const uchar* const input_green, const uchar* const input_blue,
                    uchar** output_red, uchar** output_green, uchar** output_blue)
{
  float* filter;
  get_filter_cpu(&filter);

  blur_image_cpu_channel(height, width, input_red, filter, output_red);
  blur_image_cpu_channel(height, width, input_green, filter, output_green);
  blur_image_cpu_channel(height, width, input_blue, filter, output_blue);

  free(filter);
}

void print_channel(uchar* channel, size_t height, size_t width) {
  for (size_t i = 0; i < height; ++i) {
    for (size_t j = 0; j < width; ++j) {
      std::cout << int(channel[IDX(i, j, width)]) << ' ';
    }
    std::cout << '\n';
  }
}

void assert_equal(const uchar* const cpu, const uchar* const gpu, size_t height, size_t width) {
  unsigned int diffs = 0;
  for (size_t row = 0; row < height; ++row) {
    for (size_t col = 0; col < width; ++col) {
      diffs += abs(int(cpu[IDX(row, col, width)]) - int(gpu[IDX(row, col, width)])) > 1;
    }
  }
  if (diffs != 0) {
    std::cerr << "Number of pixels with diff: " << diffs << std::endl;
    assert(false);
  }
}

int main(int argc, const char* argv[]) {
  if (argc <= 1) {
    std::cerr << "usage: ./<binary> rgb.txt [height] [width]" << std::endl;
    return 1;
  }
  freopen(argv[1], "r", stdin);

  size_t max_height = std::numeric_limits<size_t>::max();
  if (argc >= 3) {
    max_height = std::atoi(argv[2]);
  }
  size_t max_width = std::numeric_limits<size_t>::max();
  if (argc >= 4) {
    max_width = std::atoi(argv[3]);
  }

  size_t height;
  size_t width;
  std::cin >> height >> width;

  uchar* input_red;
  uchar* input_green;
  uchar* input_blue;
  read_rgb(height, width, max_height, max_width, &input_red, &input_green, &input_blue);

  if (height > max_height) height = max_height;
  if (width > max_width) width = max_width;

  std::cerr << "height: " << height << std::endl;
  std::cerr << "width: " << width << std::endl;
  std::cerr << "blur kernel size: " << BLUR_KERNEL_WIDTH << "x" << BLUR_KERNEL_WIDTH << std::endl;

  uchar* output_red_cpu;
  uchar* output_green_cpu;
  uchar* output_blue_cpu;
  {
    auto start_time = std::chrono::high_resolution_clock::now();

    blur_image_cpu(height, width, input_red, input_green, input_blue,
                   &output_red_cpu, &output_green_cpu, &output_blue_cpu);

    auto end_time = std::chrono::high_resolution_clock::now();
    auto total_time_ms = std::chrono::duration_cast<std::chrono::milliseconds>(end_time - start_time).count();
    std::cerr << "Time (CPU): " << total_time_ms / 1000.0 << " s" << std::endl;
  }

  uchar* output_red_gpu;
  uchar* output_green_gpu;
  uchar* output_blue_gpu;
  {
    auto start_time = std::chrono::high_resolution_clock::now();

    blur_image_gpu(height, width, input_red, input_green, input_blue,
                   &output_red_gpu, &output_green_gpu, &output_blue_gpu);

    auto end_time = std::chrono::high_resolution_clock::now();
    auto total_time_ms = std::chrono::duration_cast<std::chrono::milliseconds>(end_time - start_time).count();
    std::cerr << "Time (GPU): " << total_time_ms / 1000.0 << " s" << std::endl;
  }

  assert_equal(output_red_cpu, output_red_gpu, height, width);
  assert_equal(output_green_cpu, output_green_gpu, height, width);
  assert_equal(output_blue_cpu, output_blue_gpu, height, width);

  std::cout << height << ' ' << width << '\n';
  print_channel(output_red_gpu, height, width);
  print_channel(output_green_gpu, height, width);
  print_channel(output_blue_gpu, height, width);

  free(input_red);
  free(input_green);
  free(input_blue);

  free(output_red_cpu);
  free(output_green_cpu);
  free(output_blue_cpu);

  free(output_red_gpu);
  free(output_green_gpu);
  free(output_blue_gpu);
}
